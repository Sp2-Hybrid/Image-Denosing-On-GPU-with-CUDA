#include "hip/hip_runtime.h"
#include "test.h"

#include "book.h"
#include "hip/hip_runtime.h"

__global__ void  mid_process(unsigned char *dev_original,unsigned char *dev_out,int R,int C){
	int x=threadIdx.x;
	int y=blockIdx.x;
	int offset=x+y*blockDim.x;

	while(offset<R*C){
		int dev_y[NKsize*NKsize];
		int gx=offset/C;
		int gy=offset%C;
		int fx=gx-NKsize/2;
		int fy=gy-NKsize/2;
		for(int i=0;i<NKsize;i++){
			for(int j=0;j<NKsize;j++){
				if((fx+i)<0||(fy+j)<0||(fx+i)>=R||(fy+j)>=C){
					dev_y[i*NKsize+j]=dev_original[offset];
				}
				else{
					dev_y[i*NKsize+j]=dev_original[(fx+i)*C+(fy+j)];
				}
			}
		}
		int temp;
		for(int i=0;i<NKsize*NKsize-1;i++){
			for(int j=0;j<NKsize*NKsize-1-i;j++){
				temp=dev_y[j];
				dev_y[j]=dev_y[j+1];
				dev_y[j+1]=temp;
			}
		}
		dev_out[offset]=dev_y[NKsize*NKsize/2];

		offset+=blockDim.x*gridDim.x;
	}
}

extern "C" float Test_mid_GPU(string name){
	hipEvent_t start,end;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&end));

	unsigned char *original;
	unsigned char *out;
	unsigned char *dev_original;
	unsigned char *dev_out;

	Mat srcImage=imread(name);
	cvtColor(srcImage, srcImage, CV_BGR2GRAY);
	if(srcImage.channels()!=1)
		return srcImage.channels();
	Mat dstImage=srcImage.clone();

	int R= srcImage.rows;
	int C= srcImage.cols;

	original=(unsigned char*)malloc(R*C*sizeof(unsigned char));
	for(int i=0;i<R;i++){
		for(int j=0;j<C;j++){
			original[i*C+j]=srcImage.at<uchar>(i,j);
		}
	}
	out=(unsigned char*)malloc(R*C*sizeof(unsigned char));
	HANDLE_ERROR(hipMalloc((void**)&dev_original,R*C*sizeof(unsigned char)));
	HANDLE_ERROR(hipMalloc((void**)&dev_out,R*C*sizeof(unsigned char)));

	HANDLE_ERROR(hipMemcpy(dev_original,original,R*C*sizeof(unsigned char),hipMemcpyHostToDevice));

	HANDLE_ERROR(hipEventRecord(start,0));
	mid_process<<<DIM,DIM>>>(dev_original,dev_out,R,C);
	HANDLE_ERROR(hipMemcpy(out,dev_out,R*C*sizeof(unsigned char),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipEventRecord(end,0));
	HANDLE_ERROR(hipEventSynchronize(end));
	float time;
	HANDLE_ERROR(hipEventElapsedTime(&time,start,end));

	for(int i=0;i<R;i++){
		for(int j=0;j<C;j++){
			dstImage.at<uchar>(i,j)=out[i*C+j];
		}
	}
	imshow("��ֵ�˲�GPU",dstImage);

	return time;
}