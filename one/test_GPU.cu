#include "hip/hip_runtime.h"
#include "test.h"
#include "book.h"
#include "hip/hip_runtime.h"
//��ֵ�˲�GPU**************************************************************************************//
__global__ void ave_process(unsigned char* dev_original,unsigned char *dev_out,int R,int C){
	int x=threadIdx.x;
	int y=blockIdx.x;
	int offset=x+y*blockDim.x;
	
	while(offset<R*C){
		int dev_y[NKsize*NKsize];
		int gx=offset/C;
		int gy=offset%C;
		int sum=0;
		int fx=gx-NKsize/2;
		int fy=gy-NKsize/2;

		for(int i=0;i<NKsize;i++){
			for(int j=0;j<NKsize;j++){
				if((fx+i)<0||(fy+j)<0||(fx+i)>=R||(fy+j)>=C){
					dev_y[i*NKsize+j]=dev_original[offset];
				}
				else{
					dev_y[i*NKsize+j]=dev_original[(fx+i)*C+(fy+j)];
				}
				sum+=dev_y[i*NKsize+j];
			}
		}
		
		dev_out[offset]=sum/(NKsize*NKsize);
//		dev_out[offset]=dev_original[offset];

		offset+=blockDim.x*gridDim.x;
	}
}
//��ֵ�˲�GPU**************************************************************************************//
__global__ void  mid_process(unsigned char *dev_original,unsigned char *dev_out,int R,int C){
	int x=threadIdx.x;
	int y=blockIdx.x;
	int offset=x+y*blockDim.x;

	while(offset<R*C){
		int dev_y[NKsize*NKsize];
		int gx=offset/C;
		int gy=offset%C;
		int fx=gx-NKsize/2;
		int fy=gy-NKsize/2;
		for(int i=0;i<NKsize;i++){
			for(int j=0;j<NKsize;j++){
				if((fx+i)<0||(fy+j)<0||(fx+i)>=R||(fy+j)>=C){
					dev_y[i*NKsize+j]=dev_original[offset];
				}
				else{
					dev_y[i*NKsize+j]=dev_original[(fx+i)*C+(fy+j)];
				}
			}
		}
		int temp;
		for(int i=0;i<NKsize*NKsize-1;i++){
			for(int j=0;j<NKsize*NKsize-1-i;j++){
				if(dev_y[j]>dev_y[j+1]){
					temp=dev_y[j];
					dev_y[j]=dev_y[j+1];
					dev_y[j+1]=temp;
				}
			}
		}
		dev_out[offset]=dev_y[NKsize*NKsize/2];
		offset+=blockDim.x*gridDim.x;
	}
}
//����˻ع�**************************************************************************************//
float Kcal(int x1,int y1,int x0,int y0) {
	float distance=(x1-x0)*(x1-x0)+(y1-y0)*(y1-y0);
	float h=1;
	float t=distance/h;
	float xishu=1/((2*pai));

//	float zhishu=(-0.5)*((x1-x0)*(x1-x0)+(y1-y0)*(y1-y0));
	float r=xishu*exp(-0.5*t*t);
	return r;
}
void init_Wcal(float *W) {
	int offset;
	int x0=Ksize/2;
	int y0=Ksize/2;

	for(int i=0; i<Ksize*Ksize; i++) {
		for(int j=0; j<Ksize*Ksize; j++) {
			if(i==j) {
				int fx=i/Ksize;
				int fy=i%Ksize;
				W[i*Ksize*Ksize+j]=Kcal(fx,fy,x0,y0);
			} else {
				W[i*Ksize*Ksize+j]=0;
			}

		}
	}
}
//GPU�еľ���˷�����
__device__ void Mult(float *c,float* a,float *b,int a_r,int a_c,int b_r,int b_c){
	for(int i=0;i<a_r;i++){
		for(int j=0;j<b_c;j++){
			c[i*b_c+j]=0;
			for(int k=0;k<a_c;k++){
				c[i*b_c+j]+=a[i*a_c+k]*b[k*b_c+j];
			}
		}
	}
}
__global__ void cla_process(unsigned char *dev_original,unsigned char *dev_out,float *dev_mid,int R,int C){
	int x=blockIdx.x;
	int y=threadIdx.x;
	int offset=y+x*blockDim.x;
	while(offset<R*C){
	float dev_y[Ksize*Ksize];
	float dev_res[3*1];
	int fx=offset/C-Ksize/2;
	int fy=offset%C-Ksize/2;

	for(int i=0;i<Ksize;i++){
		for(int j=0;j<Ksize;j++){
			if((fx+i)<0||(fy+j)<0||(fx+i)>=R||(fy+j)>=C)
				dev_y[i*Ksize+j]=dev_original[offset];
			else
				dev_y[i*Ksize+j]=dev_original[(fx+i)*C+(fy+j)];
		}
	}
	Mult(dev_res,dev_mid,dev_y,3,Ksize*Ksize,Ksize*Ksize,1);
	float value=dev_res[0];
//	value/=1.3;
	int grayvalue=(int )value;
	if(grayvalue>255){
		grayvalue=255;
	}
	else if(grayvalue<0){
		grayvalue=0;
	}

	dev_out[offset]=(unsigned char)grayvalue;
	offset=offset+blockDim.x*gridDim.x;
	}
}
//����Ӧ�˻ع�**************************************************************************************//
__device__ float Kself(int x1,int y1,int x0,int y0,float *C){

	float h=4;
	float distance=(x1-x0)*(x1-x0)*C[0]+(x1-x0)*(y1-y0)*C[2]+(x1-x0)*(y1-y0)*C[1]+(y1-y0)*(y1-y0)*C[3];
	float xishu=sqrt(C[0]*C[3]-C[1]*C[2])/(2*pai*h*h);
	float r=xishu*exp(-0.5*distance/(h*h));
	return r;
}

//��ʼ��W����  (Ksize2 x Ksize2)
__device__ void init_Wself(float* W,float *C){
	int offset;
	int x0=Ksize/2;
	int y0=Ksize/2;

	for(int i=0;i<Ksize*Ksize;i++){
		for(int j=0;j<Ksize*Ksize;j++){
			if(i==j){
				int fx=i/Ksize;
				int fy=i%Ksize;
				W[i*Ksize*Ksize+j]=Kself(fx,fy,x0,y0,C);
			}
			else{
				W[i*Ksize*Ksize+j]=0;
			}
		}
	}
}
//��ʼ���ݶȾ���
__device__ void init_Gself(float *G,float *y){
	int fx=Ksize/2;
	int fy=Ksize/2;
	int num=0;
	for(int i=0;i<Ksize;i++){
		for(int j=0;j<Ksize;j++){
			if(i!=fx){
				G[num*2]=(y[i*Ksize+j]-y[fx*Ksize+fy])/(i-fx);
			}
			else{
				G[num*2]=0;
			}
			if(j!=fy){
				G[num*2+1]=(y[i*Ksize+j]-y[fx*Ksize+fy])/(j-fy);
			}
			else{
				G[num*2+1]=0;
			}
		/*	if(i+1>=Ksize){
				G[num*2]=0;
			}
			else{
				G[num*2]=(y[(i+1)*Ksize+j]-y[i*Ksize+j]);
			}
			if(j+1>=Ksize){
				G[num*2+1]=0;
			}
			else{
				G[num*2+1]=y[i*Ksize+j+1]-y[i*Ksize+j];
			}*/
			num++;
		}
	}
}
__device__ void pppself(float a[],float e[],float s[],float v[],int m,int n) {
	int i,j,p,q;
	double d;
	if(m>=n)
		i=n;
	else
		i=m;
	for(j=1; j<=i-1; j++) {
		a[(j-1)*n+j-1]=s[j-1];
		a[(j-1)*n+j]=e[j-1];
	}
	a[(i-1)*n+i-1]=s[i-1];
	if(m<n)
		a[(i-1)*n+i]=e[i-1];
	for(i=1; i<=n-1; i++)
		for(j=i+1; j<=n; j++) {
			p=(i-1)*n+j-1;
			q=(j-1)*n+i-1;
			d=v[p];
			v[p]=v[q];
			v[q]=d;
		}
	return;
}

__device__ void sssself(float fg[2],float cs[2]) {
	float r,d;
	//if((fabs(fg[0])+fabs(fg[1]))==0.0)
	if((fabs(fg[0])+fabs(fg[1]))<MIN_DOUBLE) {
		cs[0]=1.0;
		cs[1]=0.0;
		d=0.0;
	} else {
		d=sqrt(fg[0]*fg[0]+fg[1]*fg[1]);
		if(fabs(fg[0])>fabs(fg[1])) {
			d=fabs(d);
			if(fg[0]<0.0)
				d=-d;
		}
		if(fabs(fg[1])>=fabs(fg[0])) {
			d=fabs(d);
			if(fg[1]<0.0)
				d=-d;
		}
		cs[0]=fg[0]/d;
		cs[1]=fg[1]/d;
	}
	r=1.0;
	if(fabs(fg[0])>fabs(fg[1]))
		r=cs[1];
	else
		//if(cs[0]!=0.0)
		if(fabs(cs[0])>MIN_DOUBLE)
			r=1.0/cs[0];
	fg[0]=d;
	fg[1]=r;
	return;
}
//�Ծ����������ֵ�ֽ�
__device__ int dluavself(float a[],int m,int n,float u[],float v[],float eps,int ka){
	int i,j,k,l,it,ll,kk,ix,iy,mm,nn,iz,ml,ks;
	float d,dd,t,sm,sml,eml,sk,ek,b,c,shh,fg[2],cs[2];

	float s[26];
	float w[26];
	float e[26];

	for(i=1; i<=m; i++) {
		ix=(i-1)*m+i-1;
		u[ix]=0;
	}
	for(i=1; i<=n; i++) {
		iy=(i-1)*n+i-1;
		v[iy]=0;
	}
	it=MAX_ITERA;
	k=n;
	if(m-1<n)
		k=m-1;
	l=m;
	if(n-2<m) l=n-2;
	if(l<0) l=0;
	ll=k;
	if(l>k) ll=l;
	if(ll>=1) {
		for(kk=1; kk<=ll; kk++) {
			if(kk<=k) {
				d=0.0;
				for(i=kk; i<=m; i++) {
					ix=(i-1)*n+kk-1;
					d=d+a[ix]*a[ix];
				}
				s[kk-1]=sqrt(d);
				//if(s[kk-1]!=0.0)
				if(fabs(s[kk-1])>MIN_DOUBLE) {
					ix=(kk-1)*n+kk-1;
					//if(a[ix]!=0.0)
					if(fabs(a[ix])>MIN_DOUBLE) {
						s[kk-1]=fabs(s[kk-1]);
						if(a[ix]<0.0) s[kk-1]=-s[kk-1];
					}
					for(i=kk; i<=m; i++) {
						iy=(i-1)*n+kk-1;
						a[iy]=a[iy]/s[kk-1];
					}
					a[ix]=1.0+a[ix];
				}
				s[kk-1]=-s[kk-1];
			}
			if(n>=kk+1) {
				for(j=kk+1; j<=n; j++) {
					//if((kk<=k)&&(s[kk-1]!=0.0))
					if((kk<=k)&&(fabs(s[kk-1])>MIN_DOUBLE)) {
						d=0.0;
						for(i=kk; i<=m; i++) {
							ix=(i-1)*n+kk-1;
							iy=(i-1)*n+j-1;
							d=d+a[ix]*a[iy];
						}
						d=-d/a[(kk-1)*n+kk-1];
						for(i=kk; i<=m; i++) {
							ix=(i-1)*n+j-1;
							iy=(i-1)*n+kk-1;
							a[ix]=a[ix]+d*a[iy];
						}
					}
					e[j-1]=a[(kk-1)*n+j-1];
				}
			}
			if(kk<=k) {
				for(i=kk; i<=m; i++) {
					ix=(i-1)*m+kk-1;
					iy=(i-1)*n+kk-1;
					u[ix]=a[iy];
				}
			}
			if(kk<=l) {
				d=0.0;
				for(i=kk+1; i<=n; i++)
					d=d+e[i-1]*e[i-1];
				e[kk-1]=sqrt(d);
				//if(e[kk-1]!=0.0)
				if(fabs(e[kk-1])>MIN_DOUBLE) {
					//if(e[kk]!=0.0)
					if(fabs(e[kk])>MIN_DOUBLE) {
						e[kk-1]=fabs(e[kk-1]);
						if(e[kk]<0.0)
							e[kk-1]=-e[kk-1];
					}
					for(i=kk+1; i<=n; i++)
						e[i-1]=e[i-1]/e[kk-1];
					e[kk]=1.0+e[kk];
				}
				e[kk-1]=-e[kk-1];
				//if((kk+1<=m)&&(e[kk-1]!=0.0))
				if((kk+1<=m)&&(fabs(e[kk-1])>MIN_DOUBLE)) {
					for(i=kk+1; i<=m; i++) w[i-1]=0.0;
					for(j=kk+1; j<=n; j++)
						for(i=kk+1; i<=m; i++)
							w[i-1]=w[i-1]+e[j-1]*a[(i-1)*n+j-1];
					for(j=kk+1; j<=n; j++)
						for(i=kk+1; i<=m; i++) {
							ix=(i-1)*n+j-1;
							a[ix]=a[ix]-w[i-1]*e[j-1]/e[kk];
						}
				}
				for(i=kk+1; i<=n; i++)
					v[(i-1)*n+kk-1]=e[i-1];
			}
		}
	}
	mm=n;
	if(m+1<n) mm=m+1;
	if(k<n) s[k]=a[k*n+k];
	if(m<mm) s[mm-1]=0.0;
	if(l+1<mm) e[l]=a[l*n+mm-1];
	e[mm-1]=0.0;
	nn=m;
	if(m>n) nn=n;
	if(nn>=k+1) {
		for(j=k+1; j<=nn; j++) {
			for(i=1; i<=m; i++)
				u[(i-1)*m+j-1]=0.0;
			u[(j-1)*m+j-1]=1.0;
		}
	}
	if(k>=1) { /////////////////////////////////
		for(ll=1; ll<=k; ll++) {
			kk=k-ll+1;
			iz=(kk-1)*m+kk-1;
			//if(s[kk-1]!=0.0)
			if(fabs(s[kk-1])>MIN_DOUBLE) {
				if(nn>=kk+1)
					for(j=kk+1; j<=nn; j++) {
						d=0.0;
						for(i=kk; i<=m; i++) {
							ix=(i-1)*m+kk-1;
							iy=(i-1)*m+j-1;
							d=d+u[ix]*u[iy]/u[iz];
						}
						d=-d;
						for(i=kk; i<=m; i++) {
							ix=(i-1)*m+j-1;
							iy=(i-1)*m+kk-1;
							u[ix]=u[ix]+d*u[iy];
						}
					}
				for(i=kk; i<=m; i++) {
					ix=(i-1)*m+kk-1;
					u[ix]=-u[ix];
				}
				u[iz]=1.0+u[iz];
				if(kk-1>=1)//////////////////////////////////////
					for(i=1; i<=kk-1; i++)
						u[(i-1)*m+kk-1]=0.0;
			} else {
				for(i=1; i<=m; i++)
					u[(i-1)*m+kk-1]=0.0;
				u[(kk-1)*m+kk-1]=1.0;
			}
		}
	}
	for(ll=1; ll<=n; ll++) {
		kk=n-ll+1;
		iz=kk*n+kk-1;
		//if((kk<=l)&&(e[kk-1]!=0.0))/////////////////////////////
		if((kk<=l)&&(fabs(e[kk-1])>MIN_DOUBLE)) {
			for(j=kk+1; j<=n; j++) {
				d=0.0;
				for(i=kk+1; i<=n; i++) {
					ix=(i-1)*n+kk-1;
					iy=(i-1)*n+j-1;
					d=d+v[ix]*v[iy]/v[iz];
				}
				d=-d;
				for(i=kk+1; i<=n; i++) {
					ix=(i-1)*n+j-1;
					iy=(i-1)*n+kk-1;
					v[ix]=v[ix]+d*v[iy];
				}
			}
		}
		for(i=1; i<=n; i++)
			v[(i-1)*n+kk-1]=0.0;
		v[iz-n]=1.0;
	}
	for(i=1; i<=m; i++)
		for(j=1; j<=n; j++)
			a[(i-1)*n+j-1]=0.0;
	ml=mm;
	it=MAX_ITERA;
	while(1==1) { //////////////////////////////////
		if(mm==0) {
			pppself(a,e,s,v,m,n);

//			free(s);free(e);free(w);
			return l;
		}
		if(it==0) {
			pppself(a,e,s,v,m,n);

//			free(s);free(e);free(w);
			return -1;
		}
		kk=mm-1;
		//while((kk!=0)&&(fabs(e[kk-1])!=0.0))
		while((kk!=0)&&(fabs(e[kk-1])>MIN_DOUBLE)) {
			d=fabs(s[kk-1])+fabs(s[kk]);
			dd=fabs(e[kk-1]);
			if(dd>eps*d)
				kk=kk-1;
			else
				e[kk-1]=0.0;
		}
		if(kk==mm-1) {
			kk=kk+1;
			if(s[kk-1]<0.0) {
				s[kk-1]=-s[kk-1];
				for(i=1; i<=n; i++) {
					ix=(i-1)*n+kk-1;
					v[ix]=-v[ix];
				}
			}
			while((kk!=ml)&&(s[kk-1]<s[kk])) {
				d=s[kk-1];
				s[kk-1]=s[kk];
				s[kk]=d;
				if(kk<n)
					for(i=1; i<=n; i++) {
						ix=(i-1)*n+kk-1;
						iy=(i-1)*n+kk;
						d=v[ix];
						v[ix]=v[iy];
						v[iy]=d;
					}
				if(kk<m)
					for(i=1; i<=m; i++) {
						ix=(i-1)*m+kk-1;
						iy=(i-1)*m+kk;
						d=u[ix];
						u[ix]=u[iy];
						u[iy]=d;
					}
				kk=kk+1;
			}
			it=MAX_ITERA;
			mm=mm-1;
		} else {
			ks=mm;
			//while((ks>kk)&&(fabs(s[ks-1])!=0.0))
			while((ks>kk)&&(fabs(s[ks-1])>MIN_DOUBLE)) {
				d=0.0;
				if(ks!=mm)
					d=d+fabs(e[ks-1]);
				if(ks!=kk+1) d=d+fabs(e[ks-2]);
				dd=fabs(s[ks-1]);
				if(dd>eps*d)
					ks=ks-1;
				else
					s[ks-1]=0.0;
			}
			if(ks==kk) {
				kk=kk+1;
				d=fabs(s[mm-1]);
				t=fabs(s[mm-2]);
				if(t>d)
					d=t;
				t=fabs(e[mm-2]);
				if(t>d)
					d=t;
				t=fabs(s[kk-1]);
				if(t>d)
					d=t;
				t=fabs(e[kk-1]);
				if(t>d)
					d=t;
				sm=s[mm-1]/d;
				sml=s[mm-2]/d;
				eml=e[mm-2]/d;
				sk=s[kk-1]/d;
				ek=e[kk-1]/d;
				b=((sml+sm)*(sml-sm)+eml*eml)/2.0;
				c=sm*eml;
				c=c*c;
				shh=0.0;
				//if((b!=0.0)||(c!=0.0))
				if((fabs(b)>MIN_DOUBLE)||(fabs(c)>MIN_DOUBLE)) {
					shh=sqrt(b*b+c);
					if(b<0.0)
						shh=-shh;
					shh=c/(b+shh);
				}
				fg[0]=(sk+sm)*(sk-sm)-shh;
				fg[1]=sk*ek;
				for(i=kk; i<=mm-1; i++) {
					sssself(fg,cs);
					if(i!=kk)
						e[i-2]=fg[0];
					fg[0]=cs[0]*s[i-1]+cs[1]*e[i-1];
					e[i-1]=cs[0]*e[i-1]-cs[1]*s[i-1];
					fg[1]=cs[1]*s[i];
					s[i]=cs[0]*s[i];
					//if((cs[0]!=1.0)||(cs[1]!=0.0))
					if((fabs(cs[0]-1.0)>MIN_DOUBLE)||(fabs(cs[1])>MIN_DOUBLE))
						for(j=1; j<=n; j++) {
							ix=(j-1)*n+i-1;
							iy=(j-1)*n+i;
							d=cs[0]*v[ix]+cs[1]*v[iy];
							v[iy]=-cs[1]*v[ix]+cs[0]*v[iy];
							v[ix]=d;
						}
					sssself(fg,cs);
					s[i-1]=fg[0];
					fg[0]=cs[0]*e[i-1]+cs[1]*s[i];
					s[i]=-cs[1]*e[i-1]+cs[0]*s[i];
					fg[1]=cs[1]*e[i];
					e[i]=cs[0]*e[i];
					if(i<m)
						//if((cs[0]!=1.0)||(cs[1]!=0.0))
						if((fabs(cs[0]-1.0)>MIN_DOUBLE)||(fabs(cs[1])>MIN_DOUBLE))
							for(j=1; j<=m; j++) {
								ix=(j-1)*m+i-1;
								iy=(j-1)*m+i;
								d=cs[0]*u[ix]+cs[1]*u[iy];
								u[iy]=-cs[1]*u[ix]+cs[0]*u[iy];
								u[ix]=d;
							}
				}
				e[mm-2]=fg[0];
				it=it-1;
			} else {
				if(ks==mm) {
					kk=kk+1;
					fg[1]=e[mm-2];
					e[mm-2]=0.0;
					for(ll=kk; ll<=mm-1; ll++) {
						i=mm+kk-ll-1;
						fg[0]=s[i-1];
						sssself(fg,cs);
						s[i-1]=fg[0];
						if(i!=kk) {
							fg[1]=-cs[1]*e[i-2];
							e[i-2]=cs[0]*e[i-2];
						}
						//if((cs[0]!=1.0)||(cs[1]!=0.0))
						if((fabs(cs[0]-1.0)>MIN_DOUBLE)||(fabs(cs[1])>MIN_DOUBLE))
							for(j=1; j<=n; j++) {
								ix=(j-1)*n+i-1;
								iy=(j-1)*n+mm-1;
								d=cs[0]*v[ix]+cs[1]*v[iy];
								v[iy]=-cs[1]*v[ix]+cs[0]*v[iy];
								v[ix]=d;
							}
					}
				} else {
					kk=ks+1;
					fg[1]=e[kk-2];
					e[kk-2]=0.0;
					for(i=kk; i<=mm; i++) {
						fg[0]=s[i-1];
						sssself(fg,cs);
						s[i-1]=fg[0];
						fg[1]=-cs[1]*e[i-1];
						e[i-1]=cs[0]*e[i-1];
						//if((cs[0]!=1.0)||(cs[1]!=0.0))
						if((fabs(cs[0]-1.0)>MIN_DOUBLE)||(fabs(cs[1])>MIN_DOUBLE))
							for(j=1; j<=m; j++) {
								ix=(j-1)*m+i-1;
								iy=(j-1)*m+kk-2;
								d=cs[0]*u[ix]+cs[1]*u[iy];
								u[iy]=-cs[1]*u[ix]+cs[0]*u[iy];
								u[ix]=d;
							}
					}
				}
			}
		}
	}

			free(s);free(e);free(w);
	return l;
}
//GPU�ж�3x3����ȡ��
__device__ void Inverse(float *a){
	float A=a[0]*a[4]*a[8]+a[1]*a[5]*a[6]+a[2]*a[3]*a[7]-a[2]*a[4]*a[6]-a[1]*a[3]*a[8]-a[0]*a[5]*a[7];

	float b[9];
	b[0]=a[4]*a[8]-a[7]*a[5];
	b[1]=-1*(a[1]*a[8]-a[7]*a[2]);
	b[2]=a[1]*a[5]-a[2]*a[4];
	b[3]=a[5]*a[6]-a[8]*a[3];
	b[4]=-1*(a[2]*a[6]-a[8]*a[0]);
	b[5]=a[2]*a[3]-a[0]*a[5];
	b[6]=a[3]*a[7]-a[6]*a[4];
	b[7]=-1*(a[0]*a[7]-a[6]*a[1]);
	b[8]=a[0]*a[4]-a[1]*a[3];
	for(int i=0; i<9; i++) {
		b[i]=(float)b[i]/A;
		a[i]=b[i];
	}
}
__device__ float computeself(float *X,float *X_t,float *W,float *y){
	float WW[3*Ksize*Ksize];
	float XX[3*3];
	float YY[3*1];
	float XXWW[3*Ksize*Ksize];
	float XXy[3*1];

//�������������ռ��������

	Mult(WW,X_t,W,3,Ksize*Ksize,Ksize*Ksize,Ksize*Ksize);
	Mult(XX,WW,X,3,Ksize*Ksize,Ksize*Ksize,3);
	Inverse(XX);
	Mult(XXWW,XX,WW,3,3,3,Ksize*Ksize);
	Mult(XXy,XXWW,y,3,Ksize*Ksize,Ksize*Ksize,1);
	return (XXy[0]);
}
__constant__  float XXX[3*Ksize*Ksize];
__constant__  float XXXT[3*Ksize*Ksize];

__global__ void self_process(unsigned char *dev_original,unsigned char* dev_out,float *dev_X_t,float *dev_X,int R,int C){
	int x=threadIdx.x;
	int y=blockIdx.x;
	int offset=x+y*blockDim.x;
	while(offset<R*C){
		float dev_y[Ksize*Ksize];
		float dev_W[Ksize*Ksize*Ksize*Ksize];
		float G[Ksize*Ksize*2];
		float U[Ksize*Ksize*Ksize*Ksize];
		float V[2*2];
		float eps=0.000001;
		int fx=offset/C-Ksize/2;
		int fy=offset%C-Ksize/2;
		for(int i=0;i<Ksize;i++){
			for(int j=0;j<Ksize;j++){
				if((fx+i)<0||(fy+j)<0||(fx+i)>=R||(fy+j)>=C)
					dev_y[i*Ksize+j]=dev_original[offset];
				else
					dev_y[i*Ksize+j]=dev_original[(fx+i)*C+(fy+j)];
			}
		}
		init_Gself(G,dev_y);
		int re=dluavself(G,Ksize*Ksize,2,U,V,eps,Ksize*Ksize+1);
		float sita=atan(V[1]/V[3]);
		float gama;
		float deta;
		float C[4];
		float s1;
		float s2;

		s1=G[0];
		s2=G[3];
		gama=sqrt((s1*s2+1)/25);
		deta=(s1+1)/(s2+1);

		C[0]=gama*(deta*cos(sita)*cos(sita)+1/deta*sin(sita)*sin(sita));
		C[1]=gama*(1/deta*sin(sita)*cos(sita)-deta*cos(sita)*sin(sita));
		C[2]=C[1];
		C[3]=gama*(deta*sin(sita)*sin(sita)+1/deta*cos(sita)*cos(sita));

		init_Wself(dev_W,C);
		float value=computeself(XXX,XXXT,dev_W,dev_y);
//		value/=1.1;
		if(value>255){
			value=255;
		}
		else if(value<0){
			value=0;
		}
//		value/=1.3;
		dev_out[offset]=(unsigned char)value;
		offset=offset+gridDim.x*blockDim.x;
	}
}
//������**************************************************************************************//



extern "C" float Test_GPU(string name,int choice){
	int liR=512;
	int liC=512;

	hipEvent_t start,end;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&end));
	HANDLE_ERROR(hipEventRecord(start,0));
	unsigned char *original;
	unsigned char *out;
	unsigned char *dev_original;
	unsigned char *dev_out;
	unsigned char *temp_original;
	unsigned char *temp_out;
	temp_original=(unsigned char*)malloc(liR*liC*sizeof(unsigned char));
	temp_out=(unsigned char*)malloc(liR*liC*sizeof(unsigned char));

	float *X;
	float *X_t;
	float *W;
	float *dev_mid;
	float *XX;
	float *WW;
	float *Mid;
	float *dev_X_t;
	float *dev_X;

	Mat srcImage=imread(name);
	cvtColor(srcImage, srcImage, CV_BGR2GRAY);
	if(srcImage.channels()!=1)
		return srcImage.channels();
	Mat dstImage=srcImage.clone();

	int R= srcImage.rows;
	int C= srcImage.cols;

	original=(unsigned char*)malloc(R*C*sizeof(unsigned char));
	for(int i=0;i<R;i++){
		for(int j=0;j<C;j++){
			original[i*C+j]=srcImage.at<uchar>(i,j);
		}
	}
	out=(unsigned char*)malloc(R*C*sizeof(unsigned char));
	X=(float*)malloc(3*Ksize*Ksize*sizeof(float));
	X_t=(float*)malloc(3*Ksize*Ksize*sizeof(float));
	W=(float*)malloc(Ksize*Ksize*Ksize*Ksize*sizeof(float));
	XX=(float*)malloc(sizeof(float)*3*Ksize*Ksize);
	WW=(float*)malloc(sizeof(float)*3*Ksize*Ksize);
	Mid=(float*)malloc(sizeof(float)*3*Ksize*Ksize);
	init_X(X);
	init_X_t(X_t);
	init_Wcal(W);

	HANDLE_ERROR(hipMalloc((void**)&dev_original,R*C*sizeof(unsigned char)));
	HANDLE_ERROR(hipMalloc((void**)&dev_out,R*C*sizeof(unsigned char)));
	//for classical
	HANDLE_ERROR(hipMalloc((void**)&dev_mid,3*Ksize*Ksize*sizeof(float)));
	mult(WW,X_t,W,3,Ksize*Ksize,Ksize*Ksize,Ksize*Ksize);
	mult(XX,WW,X,3,Ksize*Ksize,Ksize*Ksize,3);
	//for self
	HANDLE_ERROR(hipMalloc((void**)&dev_X,3*Ksize*Ksize*Ksize*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_X_t,3*Ksize*Ksize*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(dev_X,X,3*Ksize*Ksize*sizeof(float),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_X_t,X_t,3*Ksize*Ksize*sizeof(float),hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(XXX),X,3*Ksize*Ksize*sizeof(float)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(XXXT),X_t,3*Ksize*Ksize*sizeof(float)));


	if(inverse(XX)!=0)
		mult(Mid,XX,WW,3,3,3,Ksize*Ksize);
	HANDLE_ERROR(hipMemcpy(dev_mid,Mid,3*Ksize*Ksize*sizeof(float),hipMemcpyHostToDevice));
	

	
	if(choice==3){
		HANDLE_ERROR(hipMemcpy(dev_original,original,R*C*sizeof(unsigned char),hipMemcpyHostToDevice));
		ave_process<<<DIM,DIM>>>(dev_original,dev_out,R,C);
		HANDLE_ERROR(hipMemcpy(out,dev_out,R*C*sizeof(unsigned char),hipMemcpyDeviceToHost));
	}
	else if(choice==4){
		HANDLE_ERROR(hipMemcpy(dev_original,original,R*C*sizeof(unsigned char),hipMemcpyHostToDevice));
		mid_process<<<DIM,DIM>>>(dev_original,dev_out,R,C);
		HANDLE_ERROR(hipMemcpy(out,dev_out,R*C*sizeof(unsigned char),hipMemcpyDeviceToHost));
	}
	else if(choice==1){
		HANDLE_ERROR(hipMemcpy(dev_original,original,R*C*sizeof(unsigned char),hipMemcpyHostToDevice));
		cla_process<<<DIM/2,DIM/2>>>(dev_original,dev_out,dev_mid,R,C);
		HANDLE_ERROR(hipMemcpy(out,dev_out,R*C*sizeof(unsigned char),hipMemcpyDeviceToHost));
	}
	else{
		int numR=(R+255)/256;
		int numC=(C+255)/256;
		for(int i=0;i<numR;i++){
			for(int j=0;j<numC;j++){
				int off=i*liR*C+j*liC;
				int tempR=liR;
				int tempC=liC;
				if(i==numR-1)
					tempR=R-liR*i;
				if(j==numC-1)
					tempC=C-liC*j;
				for(int i=0;i<tempR;i++){
					for(int j=0;j<tempC;j++){
						temp_original[i*tempC+j]=original[off+i*C+j];
					}
				}
				
				HANDLE_ERROR(hipMemcpy(dev_original,temp_original,tempR*tempC*sizeof(unsigned char),hipMemcpyHostToDevice));
				self_process<<<DIM/2,DIM/2>>>(dev_original,dev_out,dev_X_t,dev_X,tempR,tempC);
				HANDLE_ERROR(hipMemcpy(temp_out,dev_out,tempR*tempC*sizeof(unsigned char),hipMemcpyDeviceToHost));
				for(int i=0;i<tempR;i++){
					for(int j=0;j<tempC;j++){
						out[off+i*C+j]=temp_out[i*tempC+j];
					}
				}
			}
		}
	}
	HANDLE_ERROR(hipEventRecord(end,0));
	HANDLE_ERROR(hipEventSynchronize(end));
	float time;
	HANDLE_ERROR(hipEventElapsedTime(&time,start,end));

	for(int i=0;i<R;i++){
		for(int j=0;j<C;j++){
			dstImage.at<uchar>(i,j)=out[i*C+j];
		}
	}
	/*string name1=name+"classical";
	string name2=name+"self";
	string name3=name+"average";
	string name4=name+"middle";*/
	String tempname="";
	int pos;
	for(int i=0;i<name.length();i++){
		if(name[i]=='.')
		{
			pos=i;
			break;
		}
	}
	if(choice==1){
		tempname=name;
		tempname.insert(pos,"_classical_GPU");
		imshow("����˻ع�GPU",dstImage);
		imwrite(tempname,dstImage);
	}
	else if(choice==2){
		tempname=name;
		tempname.insert(pos,"_adaptive_GPU");
		imshow("����Ӧ�˻ع�GPU",dstImage);
		imwrite(tempname,dstImage);
	}
	else if(choice==3){
		tempname=name;
		tempname.insert(pos,"_mean_GPU");
		imshow("��ֵ�˲�GPU",dstImage);
		imwrite(tempname,dstImage);
	}
	else{
		tempname=name;
		tempname.insert(pos,"_median_GPU");
		imshow("��ֵ�˲�GPU",dstImage);
		imwrite(tempname,dstImage);
	}
	
	
	
//	waitKey();
	return time;
}